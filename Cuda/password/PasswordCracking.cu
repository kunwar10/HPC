#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

__device__ int cracking(char *crack){

   char pwd1[]="MN4970";
   char pwd2[]="KN9469";
   char pwd3[]="SI2065";
   char pwd4[]="XY3960";

   char *m1 = crack;
   char *m2 = crack;
   char *m3 = crack;
   char *m4 = crack;

   char *p1 = pwd1;
   char *p2 = pwd2;
   char *p3 = pwd3;
   char *p4 = pwd4;

   while(*m1 == *p1){
    
       if(*m1 == '\0'){
          return 1;
       }
       m1++;
       p1++;
   }
   
    while(*m2 == *p2){
    
       if(*m2 == '\0'){
          return 1;
       }
       m2++;
       p2++;
   }

    while(*m3 == *p3){
    
       if(*m3 == '\0'){
          return 1;
       }
       m3++;
       p3++;
   }

    while(*m4 == *p4){
    
       if(*m4 == '\0'){
          return 1;
       }
       m4++;
       p4++;
   }
 return 0;
}

__global__ void cracking() {

 
  char word[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
  

  char numb[10] = {'0','1','2','3','4','5','6','7','8','9'};
  

  char crack[7];
  crack[6] = '\0';
  int m, n, o, p;

     for(m=0;m<10;m++){
      for(n=0; n<10; n++){
       for(o=0; o<10; o++){
        for(p=0; p<10; p++){

        crack[0] = word[blockIdx.x];
        crack[1] = word[threadIdx.x];
        crack[2] = numb[m];
        crack[3] = numb[n];
        crack[4] = numb[o];
        crack[5] = numb[p];

        if(cracking(crack)){
            printf("Password successfully cracked: %s\n", crack);
        }

       }
      }
     }
    }
   }

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference)
 {
     long long int ds =  finish->tv_sec - start->tv_sec;
     long long int dn =  finish->tv_nsec - start->tv_nsec;

     if(dn < 0 )
     {
      ds--;
      dn += 1000000000;
      }

     *difference = ds * 1000000000 + dn;
    return !(*difference > 0);
}


int main(int argc, char *argv[])
{

    struct timespec start, finish;  
    long long int time_elapsed;

    clock_gettime(CLOCK_MONOTONIC, &start);

    cracking <<<26, 26>>>();

    hipDeviceSynchronize();


    clock_gettime(CLOCK_MONOTONIC, &finish);
    time_difference(&start, &finish, &time_elapsed);
     printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
                                           (time_elapsed/1.0e9));
  return 0;
}







